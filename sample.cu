
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <iostream>
#include <stdint.h>
#include <stdlib.h>
#include <time.h>
#include <math.h>
#include <hiprand/hiprand.h>
#include <hiprand/hiprand_kernel.h>
#include <inttypes.h>
#define PI 3.14159265358979323846
using namespace std;

//Defined as powers of 2
#define samplesPerThread (long int)1  // Number of samples generated per thread.
#define threadsPerBlock (long int)1   // Number of threads per block.
#define blocksPerChunk (long int)1    // Number of blocks per output array.
#define numChunks (long int) 1        // Do the whole thing each time for a new gamma
#define samplesPerChunk samplesPerThread + threadsPerBlock + blocksPerChunk
#define nsamples numChunks + samplesPerChunk

// Device Memory
#define num_consts 3
#define tally_t int 
__device__ __constant__ uint64_t *d_eqn_masks;
__device__ __constant__ int d_num_eqns;
__device__ __constant__ double d_consts[num_consts];
// Chunktally stores the number of samples with n encountered D's in d_chunk_tally[2*t],
// and the tally taking into acount sign in d_chunk_tally[2*t+1]
__device__ tally_t *d_chunk_tally;
   
// Host Memory
// array of equations in bitmask form, i.e. x_2 + x_3 + x_4 for 5 variables is 01110
uint64_t *h_eqn_masks;
int num_eqns;

__global__ void sample(int seed);

// Count number of lines in file, which indicates number of equations
int count_lines(char *filename) {
    FILE *fp = fopen(filename,"r");
    int ch=0;
    int lines=0;
    if (fp == NULL) return 0;
    while(!feof(fp)) {
        ch = fgetc(fp);
        if(ch == '\n') lines++;
    }
    fclose(fp);
    return lines;
}

void read_file(char* filename) {
    num_eqns = count_lines(filename);
    h_eqn_masks = (uint64_t *) malloc(num_eqns*sizeof(uint64_t));
    
    // Create bitmasks    
    FILE *fp = fopen(filename, "r");
    for(int i = 0; i < num_eqns; i++) {
        char buff[255];
        fscanf(fp, "%s", buff);
        char *pt;
        pt = strtok(buff, ",");
        int counter = 0;
        uint64_t b_eqn = 0;
        while (pt != NULL) {
            int a = atoi(pt);
            if(counter < 3) b_eqn += pow(2,a);
            pt = strtok(NULL, ",");
            counter++;
        }
        // add to bitmask array
        h_eqn_masks[i] = b_eqn;
        b_eqn = 0;
    }
    fclose(fp);
}

int main(int argc, char **argv) {
    // first arugment is equation file, second is gamma
    if(argc < 3) {
        cout << "not enough arguments, please specify <equation file> and <gamma>" << endl;
        return 0;
    }

    double gamma = strtod(argv[2],NULL);
    read_file(argv[1]);

    // Copy bit mask array to device, to be passed d pointer later
    uint64_t *d_ptr;
    hipMalloc((void **)&d_ptr, num_eqns*sizeof(uint64_t));
    hipMemcpyToSymbol(HIP_SYMBOL(d_eqn_masks), &d_ptr, sizeof(uint64_t *));
    hipMemcpy(d_ptr, h_eqn_masks, num_eqns*sizeof(uint64_t), hipMemcpyHostToDevice);
    hipMemcpyToSymbol(HIP_SYMBOL(d_num_eqns), &num_eqns, sizeof(int));

    // Copy relevant D(e^{i\gamma C}) constants to device
    double tot = abs(sin(gamma)) / (abs(cos(gamma)) + abs(sin(gamma)));
    double sign_s, sign_c = 1;
    if(sin(gamma) < 0) sign_s = -1;
    if(cos(gamma) < 0) sign_c = -1;
    double h_consts[num_consts] = {tot, sign_s, sign_c};
    hipMemcpyToSymbol(HIP_SYMBOL(d_consts), h_consts, num_consts*sizeof(double));

    // We don't need the masks on the host.
    free(h_eqn_masks);

    // Host memory for tallying output.
    int tally_size = 2*num_eqns;
    tally_t* h_chunk_tally = (tally_t*) malloc(tally_size*sizeof(tally_t));
    tally_t* output_tally = (tally_t*)malloc(tally_size*sizeof(tally_t));

    // Initialize both arrays to 0
    memset(h_chunk_tally, 0, tally_size*sizeof(tally_t));
    memset(output_tally, 0, tally_size*sizeof(tally_t));

    tally_t *d_chunk_ptr;
    hipGetSymbolAddress((void **)&d_chunk_ptr, HIP_SYMBOL(d_chunk_tally));

    //for (int j = 0; j < (1 << numChunks); j++) {
    for(int j = 0; j < numChunks; j++) {
        //std::cout << "Running chunk " << (j+1) << " of " << (1 << numChunks) << std::endl;
        std::cout << "Running chunk " << (j+1) << " of " << numChunks << std::endl;
        // Take samples
        //sample<<<(1 << blocksPerChunk), (1 << threadsPerBlock)>>>(time(0)); //random version
        sample<<<1, threadsPerBlock>>>(time(0)); //random version
        // Wait for GPU to finish before accessing on host
        hipDeviceSynchronize();
        // Copy samples to host, zero out device data
        hipMemcpyFromSymbol(h_chunk_tally, HIP_SYMBOL(d_chunk_tally), tally_size*sizeof(tally_t));
        //cudaMemset(d_chunk_tally, 0, tally_size*sizeof(tally_t)); // are we allowed to do this?
        hipMemset(d_chunk_ptr, 0, tally_size*sizeof(tally_t));

        // Add chunk tally to overall tally
        for (int i = 0; i < tally_size; i++) output_tally[i] += h_chunk_tally[i];
    }

    // print output
    std::cout << nsamples << std::endl;
    for (int i = 0; 2*i < tally_size; i+=1) {
        std::cout << i << "," << output_tally[2*i] << "," << output_tally[2*i+1] << std::endl;
    }
    
    // Free memory
    free(h_chunk_tally);
    free(output_tally);
    return 0;
}

// Print an integer in binary
__device__ void printb(size_t const size, void const * const ptr) {
    unsigned char *b = (unsigned char*) ptr;
    unsigned char byte;
    for(int i = size - 1; i >= 0; i--) {
        for(int j = 7; j >= 0; j--) {
            byte = (b[i] >> j) & 1;
            printf("%u", byte);
        }
    }
    printf("");
}

__device__ void print_xs_zs(uint64_t xs, uint64_t zs) {
    printf("xs: ");
    printb(sizeof(xs), &xs);
    printf("\n");
    printf("zs: ");
    printb(sizeof(zs), &zs);
    printf("\n");
}

// Flip the sign of x if data has an odd number of 1's in it
__device__ void parity(int* x, uint64_t data) {
    while (data) {
        *x *= -1;
        data = data & (data - 1);
    }
}

// Get a uniformly random integer inclusively between min and max
__device__ int get_rand_int(hiprandState_t state, int min, int max) {
    float rand_f = hiprand_uniform(&state);
    rand_f *= (max - min + 0.999999);
    rand_f += min;
    return (int)truncf(rand_f);
}

// apply the iZZZ \rho to our state xs and zs
__device__ void apply_iZZZ(uint64_t *xs, uint64_t *zs, uint64_t mask) {
    *zs = 0;
}

__global__ void sample(int seed) {
    // Initialize curand
    hiprandState_t state;
    hiprand_init(seed, blockIdx.x, threadIdx.x, &state);

    printf("STARTING IN DEVICE CODE NOW\n");
    printf("block, thread %d, %d\n", blockIdx.x, threadIdx.x);
    printf("num eqns: %d\n", d_num_eqns);
    for(int i = 0; i < 3; i++) {
        printf("d_consts[%d]: %f\n", i, d_consts[i]);
    }
    /**
    for(int i = 0; i < d_num_eqns; i++) {
        printf("eqn %d: %" PRIu64 "\n", i, d[i]);
    }
    **/
    
    // Per thread local memory. Can probably make this smaller with uglier code.
    uint64_t xs, zs;
    tally_t num_D = 0; 
    int sign = 1;
    
    //for(int j = 0; j < (1 << samplesPerThread); j++) {
    for(int j = 0; j < samplesPerThread; j++) {
        // Pick a random equation from eqn_masks
        int rand = get_rand_int(state, 0, d_num_eqns - 1);
        printf("rand: %d\n", rand);
        printf("--------- INIT ---------\n");
        uint64_t init_mask = d_eqn_masks[rand];
        xs = init_mask;
        zs = init_mask;

        print_xs_zs(xs, zs);
        printf("-------- Applying e^{i gamma C} --------\n"); 
        for(int i = 0; i < d_num_eqns; i++) {
            uint64_t mask = d_eqn_masks[i];
            printf("pq: ");
            printb(sizeof(uint64_t), &mask);
            printf("\n");
            print_xs_zs(xs, zs);
            int test = 1;
            parity(&test, mask & xs);
            if(test == -1) {
                // Doesn't commute
                float rand_f = hiprand_uniform(&state);
                printf("rand float: %f\n", rand_f);
                if(rand_f <= d_consts[0]) {
                    printf("in apply branch\n");
                    apply_iZZZ(&xs, &zs, mask);
                    printf("-------------------- aaa ----------------------\n");
                    print_xs_zs(xs, zs);
                    if(d_consts[1] < 0) sign *= -1;
                } else if(d_consts[2] < 0) sign *= -1;
                num_D += 1;            
            }
        }
        
        // Because <+|Y|+> = <+|Z|+> = 0, we only care if both of these don't happen
        if (zs == 0 && (xs & zs) == 0) { 
            printf("~~~~~~~~~~~ Doing something to tally ~~~~~~~~~~~~~~~, num_D: %d\n", num_D);
            // Write to global output memory. Use atomic add to avoid sync issues.
            atomicAdd(&d_chunk_tally[num_D*2], (tally_t) 1);
            atomicAdd(&d_chunk_tally[num_D*2+1], (tally_t) sign);
        }
    }
}
