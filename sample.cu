
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <iostream>
#include <stdint.h>
#include <stdlib.h>
#include <time.h>
#include <math.h>
#include <hiprand/hiprand.h>
#include <hiprand/hiprand_kernel.h>
#include <inttypes.h>
#define PI 3.14159265358979323846
using namespace std;

//Defined as powers of 2
#define samplesPerThread (long int)1  // Number of samples generated per thread.
#define threadsPerBlock (long int)1   // Number of threads per block.
#define blocksPerChunk (long int)1    // Number of blocks per output array.
#define numChunks (long int) 1        // Do the whole thing each time for a new gamma
#define samplesPerChunk samplesPerThread + threadsPerBlock + blocksPerChunk
#define nsamples numChunks + samplesPerChunk

// Device Memory
#define num_consts 3
#define tally_t int 
__device__ __constant__ uint64_t *d_eqn_masks;
__device__ __constant__ bool *d_sols;
__device__ __constant__ int d_num_eqns;
__device__ __constant__ double d_consts[num_consts];
// Chunktally stores the number of samples with n encountered D's in d_chunk_tally[2*t],
// and the tally taking into acount sign in d_chunk_tally[2*t+1]
__device__ tally_t *d_chunk_tally;
   
// Host Memory
uint64_t *h_eqn_masks; // array of equations in bitmask form, i.e. x_2 + x_3 + x_4 for 5 variables is 01110
bool *h_sols; // solutions to each equation, either 0 or 1
int num_eqns;

__global__ void sample(int seed);

// Count number of lines in file, which indicates number of equations
int count_lines(char *filename) {
    FILE *fp = fopen(filename,"r");
    int ch=0;
    int lines=0;
    if (fp == NULL) return 0;
    while(!feof(fp)) {
        ch = fgetc(fp);
        if(ch == '\n') lines++;
    }
    fclose(fp);
    return lines;
}

void read_file(char* filename) {
    num_eqns = count_lines(filename);
    h_eqn_masks = (uint64_t *) malloc(num_eqns*sizeof(uint64_t));
    h_sols = (bool *) malloc(num_eqns*sizeof(bool));
    
    // Create bitmasks    
    FILE *fp = fopen(filename, "r");
    for(int i = 0; i < num_eqns; i++) {
        char buff[255];
        fscanf(fp, "%s", buff);
        char *pt;
        pt = strtok(buff, ",");
        int counter = 0;
        uint64_t b_eqn = 0;
        while (pt != NULL) {
            int a = atoi(pt);
            if(counter < 3) {
                b_eqn += pow(2,a);
            } else {
                h_sols[i] = a;
            }
            pt = strtok(NULL, ",");
            counter++;
        }
        // add to bitmask array
        h_eqn_masks[i] = b_eqn;
        b_eqn = 0;
    }
    fclose(fp);
}

int main(int argc, char **argv) {
    // first arugment is equation file, second is gamma
    if(argc < 3) {
        cout << "not enough arguments, please specify <equation file> and <gamma>" << endl;
        return 0;
    }

    double gamma = strtod(argv[2],NULL);
    read_file(argv[1]);

    // Copy bit mask array to device
    uint64_t *d_ptr;
    hipMalloc((void **)&d_ptr, num_eqns*sizeof(uint64_t));
    hipMemcpyToSymbol(HIP_SYMBOL(d_eqn_masks), &d_ptr, sizeof(uint64_t *));
    hipMemcpy(d_ptr, h_eqn_masks, num_eqns*sizeof(uint64_t), hipMemcpyHostToDevice);
    // Copy solutions to equations to device
    bool *sol_ptr;
    hipMalloc((void **)&sol_ptr, num_eqns*sizeof(bool));
    hipMemcpyToSymbol(HIP_SYMBOL(d_sols), &sol_ptr, sizeof(bool *));
    hipMemcpy(sol_ptr, h_sols, num_eqns*sizeof(bool), hipMemcpyHostToDevice);
    // Copy num equations to device
    hipMemcpyToSymbol(HIP_SYMBOL(d_num_eqns), &num_eqns, sizeof(int));
    // Malloc space for d_chunk_tally
    tally_t *t_ptr;
    hipMalloc((void **)&t_ptr, 2*num_eqns*sizeof(tally_t));
    hipMemcpyToSymbol(HIP_SYMBOL(d_chunk_tally), &t_ptr, sizeof(tally_t *));
    // Copy relevant D(e^{i\gamma C}) constants to device
    double tot = abs(sin(gamma)) / (abs(cos(gamma)) + abs(sin(gamma)));
    double sign_s = 1;
    double sign_c = 1;
    if(sin(gamma) < 0) sign_s = -1;
    if(cos(gamma) < 0) sign_c = -1;
    double h_consts[num_consts] = {tot, sign_s, sign_c};
    hipMemcpyToSymbol(HIP_SYMBOL(d_consts), h_consts, num_consts*sizeof(double));

    // We don't need the masks or sols on the host.
    free(h_eqn_masks);
    free(h_sols);

    // Host memory for tallying output.
    int tally_size = 2*num_eqns;
    tally_t* h_chunk_tally = (tally_t*) malloc(tally_size*sizeof(tally_t));
    tally_t* output_tally = (tally_t*)malloc(tally_size*sizeof(tally_t));

    // Initialize both arrays to 0
    memset(h_chunk_tally, 0, tally_size*sizeof(tally_t));
    memset(output_tally, 0, tally_size*sizeof(tally_t));

    //tally_t *d_chunk_ptr;
    //cudaGetSymbolAddress((void **)&d_chunk_ptr, d_chunk_tally);

    //for (int j = 0; j < (1 << numChunks); j++) {
    for(int j = 0; j < numChunks; j++) {
        //std::cout << "Running chunk " << (j+1) << " of " << (1 << numChunks) << std::endl;
        std::cout << "Running chunk " << (j+1) << " of " << numChunks << std::endl;
        // Take samples
        //sample<<<(1 << blocksPerChunk), (1 << threadsPerBlock)>>>(time(0)); //random version
        sample<<<1, threadsPerBlock>>>(time(0)); //random version
        // Wait for GPU to finish before accessing on host
        hipDeviceSynchronize();
        // Copy samples to host, zero out device data
        //cudaMemcpyFromSymbol(h_chunk_tally, d_chunk_tally, tally_size*sizeof(tally_t));
        hipMemcpy(h_chunk_tally, t_ptr, tally_size*sizeof(tally_t), hipMemcpyDeviceToHost);
        //for(int i = 0; i < tally_size; i++) {
        //    printf("tally[%d]: %d\n", i, h_chunk_tally[i]);
        //}
        //cudaMemset(d_chunk_tally, 0, tally_size*sizeof(tally_t)); // are we allowed to do this?
        hipMemset(t_ptr, 0, tally_size*sizeof(tally_t));

        // Add chunk tally to overall tally
        for (int i = 0; i < tally_size; i++) output_tally[i] += h_chunk_tally[i];
    }

    // print output
    std::cout << nsamples << std::endl;
    for (int i = 0; 2*i < tally_size; i+=1) {
        std::cout << i << "," << output_tally[2*i] << "," << output_tally[2*i+1] << std::endl;
    }
    
    // Free memory
    free(h_chunk_tally);
    free(output_tally);
    return 0;
}

// Print an integer in binary
__device__ void printb(size_t const size, void const * const ptr) {
    unsigned char *b = (unsigned char*) ptr;
    unsigned char byte;
    for(int i = size - 1; i >= 0; i--) {
        for(int j = 7; j >= 0; j--) {
            byte = (b[i] >> j) & 1;
            if(i < 2) printf("%u", byte);
            //printf("%u", byte);
        }
    }
    printf("");
}

__device__ void print_xs_zs(uint64_t xs, uint64_t zs) {
    printf("xs: ");
    printb(sizeof(xs), &xs);
    printf("\n");
    printf("zs: ");
    printb(sizeof(zs), &zs);
    printf("\n");
}

// Flip the sign of x if data has an odd number of 1's in it
__device__ void parity(int* x, uint64_t data) {
    while (data) {
        *x *= -1;
        data = data & (data - 1);
    }
}

// Get a uniformly random integer inclusively between min and max
__device__ int get_rand_int(hiprandState_t state, int min, int max) {
    float rand_f = hiprand_uniform(&state);
    rand_f *= (max - min + 0.999999);
    rand_f += min;
    return (int)truncf(rand_f);
}

__global__ void sample(int seed) {
    // Initialize curand
    hiprandState_t state;
    hiprand_init(seed, blockIdx.x, threadIdx.x, &state);

    printf("STARTING IN DEVICE CODE NOW\n");
    printf("block, thread %d, %d\n", blockIdx.x, threadIdx.x);
    printf("num eqns: %d\n", d_num_eqns);
    for(int i = 0; i < 3; i++) {
        printf("d_consts[%d]: %f\n", i, d_consts[i]);
    }
    for(int i = 0; i < d_num_eqns; i++) {
        printf("eqn[%d]: ", i);
        printb(sizeof(uint64_t), &d_eqn_masks[i]);
        printf("\n");
    }
    for(int i = 0; i < d_num_eqns; i++) {
        printf("sols[%d]: %d\n", i, d_sols[i]);
    }
    
    // Per thread local memory. Can probably make this smaller with uglier code.
    uint64_t xs, zs;
    tally_t num_D = 0; 
    int sign = 1;
    
    //for(int j = 0; j < (1 << samplesPerThread); j++) {
    for(int j = 0; j < samplesPerThread; j++) {
        // Pick a random equation from eqn_masks
        int rand = get_rand_int(state, 0, d_num_eqns - 1);
        printf("rand: %d\n", rand);
        printf("--------- INIT ---------\n");
        uint64_t init_mask = d_eqn_masks[rand];
        xs = init_mask;
        zs = init_mask;

        print_xs_zs(xs, zs);
        printf("-------- Applying e^{i gamma C} --------\n"); 
        for(int i = 0; i < d_num_eqns; i++) {
            uint64_t mask = d_eqn_masks[i];
            printf("pq: ");
            printb(sizeof(uint64_t), &mask);
            printf("\n");
            print_xs_zs(xs, zs);
            int test = 1;
            parity(&test, mask & xs);
            printf("parity: %d\n", test);
            if(test == -1) {
                // Doesn't commute
                float rand_f = hiprand_uniform(&state);
                printf("rand float: %f\n", rand_f);
                if(rand_f <= d_consts[0]) {
                    // Apply ZZZ
                    zs ^= mask;
                    // check for 3 X's (or is it 1 X?, if 1 X then change != to == below)
                    if(xs & mask & ((xs & mask) - 1) != 0) {
                        printf("3 x's\n");
                        // we have exactly 3 X's in equation region
                        sign *= -1;
                    }
                    sign *= 2*d_sols[i] - 1; // dabc
                    sign *= d_consts[1]; // d_consts[1] is sign(sin(gamma))
                } else {
                    sign *= d_consts[2]; // d_consts[1] is sign(cos(gamma))
                }
                num_D += 1;            
            }
            printf("sign: %d\n", sign);
            printf("---------------\n");
        }
        // Because <+|Y|+> = <+|Z|+> = 0, we only care if both of these don't happen
        if (zs == 0 && (xs & zs) == 0) { 
            // Write to global output memory. Use atomic add to avoid sync issues.
            atomicAdd(&d_chunk_tally[num_D*2], (tally_t) 1);
            atomicAdd(&d_chunk_tally[num_D*2+1], (tally_t) sign);
        }
    }
}
