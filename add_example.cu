
#include <hip/hip_runtime.h>
#include <stdio.h>
#define N 3

__device__ int *d;

__global__ void add() {
    printf("IN DEVICE CODE\n");
    for(int i = 0; i < 3; i++) {
        printf("d[%d]: %d\n", i, d[i]);
    }
}

int main() {
    int *h = (int *) malloc(N*sizeof(int));
    h[0] = 3;
    h[1] = 2;
    h = {0};
    for (int i = 0; i<N; ++i) {
        //h[i] = i+1;
        printf("h[%d]: %d\n", i, h[i]);
    }

    int *d_ptr;
    hipGetSymbolAddress((void **)&d_ptr, HIP_SYMBOL(d));

    //printf("%p\n", (void *) d_ptr);

    hipMalloc((void **) &d_ptr, N*sizeof(int));

    //void *d_ptr = fixed_cudaMalloc(N*sizeof(int));

    //cudaMemcpyToSymbol(d, h, N*sizeof(int));
    hipMemcpy(d_ptr, h, N*sizeof(int), hipMemcpyHostToDevice);
    free(h);

    add<<<1, 1>>>();
    hipDeviceSynchronize();

    printf("returned\n");
    //cudaFree(d_ptr);

    return 0;
}
